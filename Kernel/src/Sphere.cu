#include "hip/hip_runtime.h"
#include "Sphere.cuh"
__device__ Sphere::Sphere() {}
__device__ Sphere::Sphere(const glm::vec3& center, float radius)
  : mCenter(center)
  , mRadius(radius)
{
}

__device__ bool Sphere::hit(const Ray& r, float tMin, float tMax, HitRecord& record) const
{
  glm::vec3 oc = r.origin() - mCenter;
    float a = glm::dot(r.direction(), r.direction());
    float b = glm::dot(oc, r.direction());
    float c = glm::dot(oc, oc) - mRadius*mRadius;
    float discriminant = b*b - a*c;
    if (discriminant > 0) {
        float temp = (-b - sqrt(discriminant))/a;
        if (temp < tMax && temp > tMin) {
            record.t = temp;
            record.point = r.pointAtParameter(record.t);
            record.normal = (record.point - mCenter) / mRadius;
            return true;
        }
        temp = (-b + sqrt(discriminant)) / a;
        if (temp < tMax && temp > tMin) {
            record.t = temp;
            record.point = r.pointAtParameter(record.t);
            record.normal = (record.point - mCenter) / mRadius;
            return true;
        }
    }
    return false;
}
