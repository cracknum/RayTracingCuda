#include "hip/hip_runtime.h"
#include "Ray.cuh"
#include "RayTracer.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
namespace Kernel
{
__device__ bool hitSphere(const glm::vec3& center, float radius, const Ray& r)
{
  glm::vec3 oc = r.origin() - center;
  float a = dot(r.direction(), r.direction());
  float b = 2.0f * dot(oc, r.direction());
  float c = dot(oc, oc) - radius * radius;
  float discriminant = b * b - 4.0f * a * c;
  return (discriminant > 0.0f);
}

__device__ glm::vec3 color(const Ray& r)
{
  if (hitSphere(glm::vec3(0, 0, -1), 0.5, r))
  {
    return glm::vec3(1.0f, 0.0f, 0.0f);
  }

  glm::vec3 uDirection = glm::normalize(r.direction());
  float t = 0.5f * (uDirection.y + 1.0f);

  return (1 - t) * glm::vec3(1.0f, 1.0f, 1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);
}

__global__ void renderInternal(
  ImageInfo imageInfo, SpaceImageInfo spaceImageInfo, glm::vec3 rayOrigin)
{
  int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
  int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

  if (xIndex >= imageInfo.width || yIndex >= imageInfo.height)
  {
    return;
  }

  int pixelIndex = (yIndex * imageInfo.width + xIndex) * 3;

  float x = static_cast<float>(xIndex) / static_cast<float>(imageInfo.width);
  float y = static_cast<float>(yIndex) / static_cast<float>(imageInfo.height);

  Ray ray(rayOrigin,
    spaceImageInfo.mLowerLeftCorner + x * spaceImageInfo.mHorizontal +
      y * spaceImageInfo.mVertical);
  glm::vec3 c = color(ray);
  imageInfo.mColor[pixelIndex] = c.x * 255;
  imageInfo.mColor[pixelIndex + 1] = c.y * 255;
  imageInfo.mColor[pixelIndex + 2] = c.z * 255;
}

void RayTracer::render(
  ImageInfo& imageInfo, const SpaceImageInfo& spaceImageInfo, const glm::vec3& rayOrigin)
{
  unsigned char* devId;
  int imageSize = imageInfo.width * imageInfo.height * sizeof(unsigned char) * 3;
  hipMalloc(&devId, imageSize);
  hipMemset(devId, 0, imageSize);
  hipMemcpy(devId, imageInfo.mColor, imageSize, hipMemcpyHostToDevice);

  ImageInfo cImageInfo = imageInfo;
  cImageInfo.mColor = devId;
  dim3 threadSize(8, 8, 1);
  dim3 blockSize((imageInfo.width + 7) / 8, (imageInfo.height + 7) / 8, 1);

  renderInternal<<<blockSize, threadSize>>>(cImageInfo, spaceImageInfo, rayOrigin);

  hipDeviceSynchronize();
  hipMemcpy(imageInfo.mColor, cImageInfo.mColor, imageSize, hipMemcpyDeviceToHost);

  hipFree(cImageInfo.mColor);
}

struct RayTracer::Impl
{
  hipGraphicsResource* mPBOResource;
  unsigned char* mImageDeviceId;
  size_t mResourceSize;
  Impl()
    : mPBOResource(nullptr)
    , mImageDeviceId(nullptr)
    , mResourceSize(0)
  {
  }
};

void RayTracer::bindImagePBO(GLuint pbo)
{
  hipGraphicsGLRegisterBuffer(&mImpl->mPBOResource, pbo, cudaGLMapFlagsWriteDiscard);
}

void RayTracer::unbindImagePBO(GLuint pbo)
{
  hipGraphicsUnregisterResource(mImpl->mPBOResource);
}
void RayTracer::updateImage(ImageInfo& imageInfo, const SpaceImageInfo& spaceImageInfo, const glm::vec3& rayOrigin)
{
  if (!mImpl->mPBOResource)
  {
    return;
  }

  hipGraphicsMapResources(1, &mImpl->mPBOResource, nullptr);

  hipGraphicsResourceGetMappedPointer(
    reinterpret_cast<void**>(&mImpl->mImageDeviceId), &mImpl->mResourceSize, mImpl->mPBOResource);

  int imageSize = imageInfo.width * imageInfo.height * sizeof(unsigned char) * 3;

  ImageInfo cImageInfo = imageInfo;
  cImageInfo.mColor = mImpl->mImageDeviceId;
  dim3 threadSize(8, 8, 1);
  dim3 blockSize((imageInfo.width + 7) / 8, (imageInfo.height + 7) / 8, 1);

  renderInternal<<<blockSize, threadSize>>>(cImageInfo, spaceImageInfo, rayOrigin);

  hipDeviceSynchronize();

  hipGraphicsUnmapResources(1, &mImpl->mPBOResource, nullptr);

}

RayTracer::RayTracer()
{
  mImpl = std::make_unique<Impl>();
}

RayTracer::~RayTracer()
{
  if (mImpl->mPBOResource)
  {
    hipGraphicsUnmapResources(1, &mImpl->mPBOResource, nullptr);
    hipGraphicsUnregisterResource(mImpl->mPBOResource);
    mImpl->mPBOResource = nullptr;
    mImpl->mImageDeviceId = nullptr;
    mImpl->mResourceSize = 0;
  }
}
} // namespace Kernel
