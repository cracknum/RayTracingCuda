#include "hip/hip_runtime.h"
#include "Camera.cuh"
#include "Hitable.cuh"
#include "HitableList.cuh"
#include "Ray.cuh"
#include "RayTracer.cuh"

#include "Dispatcher.hpp"
#include "Sphere.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <iostream>
namespace Kernel
{
__device__ bool hitSphere(const glm::vec3& center, float radius, const Ray& r)
{
  glm::vec3 oc = r.origin() - center;
  float a = dot(r.direction(), r.direction());
  float b = 2.0f * dot(oc, r.direction());
  float c = dot(oc, oc) - radius * radius;
  float discriminant = b * b - 4.0f * a * c;
  return (discriminant > 0.0f);
}

__device__ glm::vec3 color(const Ray& r, Hitable** dWorld)
{
  HitRecord record;
  if ((*dWorld)->hit(r, 0, FLT_MAX, record))
  {
    return 0.5f * glm::vec3(record.normal.x + 1.0f, record.normal.y + 1.0f, record.normal.z + 1.0f);
  }
  else
  {
    glm::vec3 uDirection = glm::normalize(r.direction());
    float t = 0.5f * (uDirection.y + 1.0f);

    return (1 - t) * glm::vec3(1.0f, 1.0f, 1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);
  }
}

__global__ void createWorld(Hitable** dList, Hitable** dWorld)
{
  *dList = new Sphere(glm::vec3(0, 0, -1), 0.5);
  *(dList + 1) = new Sphere(glm::vec3(0, -100.5, -1), 100);
  *dWorld = new HitableList(dList, 2);
}

__global__ void destroyWorld(Hitable** dList, Hitable** dWorld)
{
  delete *dList;
  delete *(dList + 1);
  delete *dWorld;
}

__global__ void renderInternal(Camera camera, ImageInfo imageInfo, glm::vec3 rayOrigin,
  Hitable** dWorld, int nsize, hiprandState* states)
{
  int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
  int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

  if (xIndex >= imageInfo.width || yIndex >= imageInfo.height)
  {
    return;
  }

  int pixelIndex = (yIndex * imageInfo.width + xIndex) * 3;
  int randIndex = yIndex * imageInfo.width + xIndex;

  glm::vec3 c(0, 0, 0);

  for (int i = 0; i < nsize; ++i)
  {
    float x = (xIndex + hiprand_uniform(&states[randIndex])) / imageInfo.width;
    float y = (yIndex + hiprand_uniform(&states[randIndex])) / imageInfo.height;
    Ray ray = camera.getRay(x, y);
    c += color(ray, dWorld);
  }

  c /= static_cast<float>(nsize);

  imageInfo.mColor[pixelIndex] = c.x * 255;
  imageInfo.mColor[pixelIndex + 1] = c.y * 255;
  imageInfo.mColor[pixelIndex + 2] = c.z * 255;
}

__global__ void initRandom(int width, int height, hiprandState* states)
{
  int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

  int index = yIndex * width + xIndex;

  if (xIndex >= width || yIndex >= height)
  {
    return;
  }

  hiprand_init(1984, index, 0, &states[index]);
}

struct RayTracer::Impl
{
  hipGraphicsResource* mPBOResource;
  unsigned char* mImageDeviceId;
  size_t mResourceSize;
  std::shared_ptr<Camera> mCamera;
  Hitable** dList;
  Hitable** dWorld;
  hiprandState* d_rand_state;
  ImageInfo mImageInfo;

  Impl()
    : mPBOResource(nullptr)
    , mImageDeviceId(nullptr)
    , mResourceSize(0)
    , d_rand_state(nullptr)
  {
    mCamera = std::make_shared<Camera>(glm::vec3(0.0f, 0.0f, -1.0f), glm::vec3(0.0f, 0.0f, 0.0f),
      glm::vec3(0.0f, 1.0f, 0.0f), 90.0f, 1920.0 / 1080);

    hipMalloc(&dList, sizeof(Hitable*) * 2);
    hipMalloc(&dWorld, sizeof(Hitable*));
    createWorld<<<1, 1>>>(dList, dWorld);
  }
  ~Impl()
  {
    destroyWorld<<<1, 1>>>(dList, dWorld);
    hipFree(dList);
    hipFree(dWorld);
  }
};

void RayTracer::bindImagePBO(GLuint pbo)
{
  hipGraphicsGLRegisterBuffer(&mImpl->mPBOResource, pbo, cudaGLMapFlagsWriteDiscard);
}

void RayTracer::unbindImagePBO(GLuint pbo)
{
  hipGraphicsUnregisterResource(mImpl->mPBOResource);
}
void RayTracer::updateImage(const ImageInfo& imageInfo)
{
  if (!mImpl->mPBOResource)
  {
    return;
  }

  hipGraphicsMapResources(1, &mImpl->mPBOResource, nullptr);
  hipGraphicsResourceGetMappedPointer(
    reinterpret_cast<void**>(&mImpl->mImageDeviceId), &mImpl->mResourceSize, mImpl->mPBOResource);

  ImageInfo cImageInfo = imageInfo;
  cImageInfo.mColor = mImpl->mImageDeviceId;
  dim3 blockSize(8, 8, 1);
  dim3 gridSize((imageInfo.width + 7) / 8, (imageInfo.height + 7) / 8, 1);
  if (imageInfo.width != mImpl->mImageInfo.width || imageInfo.height != mImpl->mImageInfo.height)
  {
    float aspect = imageInfo.width * 1.0f / imageInfo.height;
    mImpl->mCamera->setAspect(aspect);
    hipMalloc(&mImpl->d_rand_state, sizeof(hiprandState) * imageInfo.width * imageInfo.height);
    initRandom<<<gridSize, blockSize>>>(imageInfo.width, imageInfo.height, mImpl->d_rand_state);
    mImpl->mImageInfo = imageInfo;
  }
  // 抗锯齿参数
  int nSize = 1;

  renderInternal<<<gridSize, blockSize>>>(*mImpl->mCamera, cImageInfo,
    mImpl->mCamera->getCameraOrigin(), mImpl->dWorld, nSize, mImpl->d_rand_state);

  hipDeviceSynchronize();
  hipGraphicsUnmapResources(1, &mImpl->mPBOResource, nullptr);
}
Dispatcher::ObserverPtr RayTracer::getCamera() const
{
  return mImpl->mCamera;
}

RayTracer::RayTracer()
{
  mImpl = std::make_unique<Impl>();
}

RayTracer::~RayTracer()
{
  if (mImpl->mPBOResource)
  {
    hipGraphicsUnmapResources(1, &mImpl->mPBOResource, nullptr);
    hipGraphicsUnregisterResource(mImpl->mPBOResource);
    mImpl->mPBOResource = nullptr;
    mImpl->mImageDeviceId = nullptr;
    mImpl->mResourceSize = 0;
  }
}
} // namespace Kernel
