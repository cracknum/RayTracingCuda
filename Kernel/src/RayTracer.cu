#include "hip/hip_runtime.h"
#include "Ray.cuh"
#include "RayTracer.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "Hitable.cuh"
#include "Sphere.cuh"
#include "HitableList.cuh"
namespace Kernel
{
__device__ bool hitSphere(const glm::vec3& center, float radius, const Ray& r)
{
  glm::vec3 oc = r.origin() - center;
  float a = dot(r.direction(), r.direction());
  float b = 2.0f * dot(oc, r.direction());
  float c = dot(oc, oc) - radius * radius;
  float discriminant = b * b - 4.0f * a * c;
  return (discriminant > 0.0f);
}

__device__ glm::vec3 color(const Ray& r, Hitable **dWorld)
{
  HitRecord record;
  if ((*dWorld)->hit(r, 0, FLT_MAX, record))
  {
    return 0.5f*glm::vec3(record.normal.x+1.0f, record.normal.y+1.0f, record.normal.z+1.0f);
  }
  else
  {
    glm::vec3 uDirection = glm::normalize(r.direction());
    float t = 0.5f * (uDirection.y + 1.0f);

    return (1 - t) * glm::vec3(1.0f, 1.0f, 1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);
  }
}

__global__ void createWorld(Hitable** dList, Hitable** dWorld)
{
  *dList = new Sphere(glm::vec3(0, 0, -1), 0.5);
  *(dList + 1) = new Sphere(glm::vec3(0, -100.5, -1), 100);
  *dWorld = new HitableList(dList, 2);
}

__global__ void destroyWorld(Hitable** dList, Hitable** dWorld)
{
  delete *dList;
  delete *(dList + 1);
  delete *dWorld;
}

__global__ void renderInternal(
  ImageInfo imageInfo, SpaceImageInfo spaceImageInfo, glm::vec3 rayOrigin, Hitable** dWorld)
{
  int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
  int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

  if (xIndex >= imageInfo.width || yIndex >= imageInfo.height)
  {
    return;
  }

  int pixelIndex = (yIndex * imageInfo.width + xIndex) * 3;

  float x = static_cast<float>(xIndex) / static_cast<float>(imageInfo.width);
  float y = static_cast<float>(yIndex) / static_cast<float>(imageInfo.height);



  Ray ray(rayOrigin,
    spaceImageInfo.mLowerLeftCorner + x * spaceImageInfo.mHorizontal +
      y * spaceImageInfo.mVertical);
  glm::vec3 c = color(ray, dWorld);

  imageInfo.mColor[pixelIndex] = c.x * 255;
  imageInfo.mColor[pixelIndex + 1] = c.y * 255;
  imageInfo.mColor[pixelIndex + 2] = c.z * 255;
}



struct RayTracer::Impl
{
  hipGraphicsResource* mPBOResource;
  unsigned char* mImageDeviceId;
  size_t mResourceSize;
  Impl()
    : mPBOResource(nullptr)
    , mImageDeviceId(nullptr)
    , mResourceSize(0)
  {
  }
};

void RayTracer::bindImagePBO(GLuint pbo)
{
  hipGraphicsGLRegisterBuffer(&mImpl->mPBOResource, pbo, cudaGLMapFlagsWriteDiscard);
}

void RayTracer::unbindImagePBO(GLuint pbo)
{
  hipGraphicsUnregisterResource(mImpl->mPBOResource);
}
void RayTracer::updateImage(ImageInfo& imageInfo, const SpaceImageInfo& spaceImageInfo, const glm::vec3& rayOrigin)
{
  if (!mImpl->mPBOResource)
  {
    return;
  }

  hipGraphicsMapResources(1, &mImpl->mPBOResource, nullptr);
  hipGraphicsResourceGetMappedPointer(
    reinterpret_cast<void**>(&mImpl->mImageDeviceId), &mImpl->mResourceSize, mImpl->mPBOResource);

  Hitable **dList;
  hipMalloc(&dList, sizeof(Hitable*) * 2);
  Hitable **dWorld;
  hipMalloc(&dWorld, sizeof(Hitable*));

  createWorld<<<1,1>>>(dList, dWorld);

  ImageInfo cImageInfo = imageInfo;
  cImageInfo.mColor = mImpl->mImageDeviceId;
  dim3 blockSize(8, 8, 1);
  dim3 gridSize((imageInfo.width + 7) / 8, (imageInfo.height + 7) / 8, 1);

  renderInternal<<<gridSize, blockSize>>>(cImageInfo, spaceImageInfo, rayOrigin, dWorld);
  auto error = hipGetLastError();
  std::cout << __LINE__ << hipGetErrorName(error) << ": " << hipGetErrorString(error) << std::endl;
  hipDeviceSynchronize();
  destroyWorld<<<1,1>>>(dList, dWorld);
  hipFree(dList);
  hipFree(dWorld);
  hipGraphicsUnmapResources(1, &mImpl->mPBOResource, nullptr);

}

RayTracer::RayTracer()
{
  mImpl = std::make_unique<Impl>();
}

RayTracer::~RayTracer()
{
  if (mImpl->mPBOResource)
  {
    hipGraphicsUnmapResources(1, &mImpl->mPBOResource, nullptr);
    hipGraphicsUnregisterResource(mImpl->mPBOResource);
    mImpl->mPBOResource = nullptr;
    mImpl->mImageDeviceId = nullptr;
    mImpl->mResourceSize = 0;
  }
}
} // namespace Kernel
