#include "hip/hip_runtime.h"
#include "ImageMaker.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

namespace MODULE_NAME
{
    namespace kernelCode
    {
        __global__ void makeImage(int width, int height, unsigned char *imageData)
        {
            int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
            int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

            if (xIndex >= width || yIndex >= height)
            {
                return;
            }

            float x = static_cast<float>(xIndex) / width;
            float y = static_cast<float>(yIndex) / height;

            int pixelIndex = (width * yIndex + xIndex) * 3;

            unsigned char red = static_cast<unsigned char>(x * 255);
            unsigned char blue = static_cast<unsigned char>(y * 255);
            unsigned char green = static_cast<unsigned char>(x * y * 255);

            imageData[pixelIndex] = red;
            imageData[pixelIndex + 1] = blue;
            imageData[pixelIndex + 2] = green;
        }

    }
    ImageMaker::ImageMaker()
    :hostId(nullptr)
    {
    }

    ImageMaker::~ImageMaker()
    {
        if (hostId)
        {
            hipHostFree(hostId);
            hostId = nullptr;
        }
        
    }

    void ImageMaker::makeImage(int width, int height)
    {
        unsigned char *devId;

        hipHostMalloc(&hostId, width * height * 3);
        memset(hostId, 0, sizeof(unsigned char) * width * height * 3);
        hipMalloc(&devId, width * height * 3);
        hipMemset(devId, 0, sizeof(unsigned char) * width * height * 3);

        dim3 threadBlock(8, 8, 1);
        dim3 gridBlock((width + 7) / 8, (height + 7) / 8, 1);
        kernelCode::makeImage<<<gridBlock, threadBlock>>>(width, height, devId);

        hipDeviceSynchronize();
        hipMemcpy(hostId, devId, sizeof(unsigned char) * width * height * 3, hipMemcpyDeviceToHost);

        hipFree(devId);
    }
}
